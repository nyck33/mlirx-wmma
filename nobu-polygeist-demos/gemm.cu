//cgeist gemm.cu -function=matmul -S --resource-dir=$LLVM_BUILD_DIR/lib/clang/18 --cuda-gpu-arch=sm_75
//--cuda-path=/usr/local/cuda-11.8 -emit-cuda
//set env var
//export LLVM_BUILD_DIR=/mnt/d/LLVM/NewPolygeistDir/llvm-project/build
#include <hip/hip_runtime.h>

#define N 200
#define M 300
#define K 400
#define DATA_TYPE float

__global__ void matmul_kernel(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        DATA_TYPE sum = 0;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * M + col];
        }
        C[row * M + col] = sum;
    }
}

void matmul(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C) {
    DATA_TYPE *d_A, *d_B, *d_C;

    // Allocate memory on the GPU
    hipMalloc((void **)&d_A, N * K * sizeof(DATA_TYPE));
    hipMalloc((void **)&d_B, K * M * sizeof(DATA_TYPE));
    hipMalloc((void **)&d_C, N * M * sizeof(DATA_TYPE));

    // Copy data from host to device
    hipMemcpy(d_A, A, N * K * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * M * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    matmul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(C, d_C, N * M * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
